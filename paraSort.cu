#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <assert.h>

//Compile
//nvcc paraSort.cu -o DQ


// Function that catches the error 
void testCUDA(hipError_t error, const char *file, int line)  {
	if (error != hipSuccess) {
	   printf("There is an error in file %s at line %d\n", file, line);
       exit(EXIT_FAILURE);
	} 
}

// Fonction pour comparer deux entiers (utilisée par qsort)
int compare(const void *a, const void *b) {
    return (*(int*)a - *(int*)b);
}

// Fonction pour générer et retourner un tableau de n valeurs aléatoires triées
int* generateSortedRandomArray(int n) {
    int *arr = (int*)malloc(n * sizeof(int));
    if (!arr) {
        printf("Erreur d'allocation de mémoire.\n");
        exit(1);
    }

    // Remplir le tableau avec des valeurs aléatoires
    for (int i = 0; i < n; i++) {
        arr[i] = rand() % (n*10);  // Génère des nombres entre 0 et 999
    }

    // Trier le tableau
    qsort(arr, n, sizeof(int), compare);

    return arr;
}

bool isSorted(int arr[], int n) {
    for (int i = 1; i < n; i++) {
        if (arr[i-1] > arr[i]) {
            return false; // Si un élément précédent est supérieur à un élément suivant, le tableau n'est pas trié
        }
    }
    return true; // Si nous avons parcouru tout le tableau sans trouver d'éléments non triés
}


// Has to be defined in the compilation in order to get the correct value 
// of the macros __FILE__ and __LINE__
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))

__global__ void mergeSmall_k(int* A ,int sizeA, int* B,int sizeB, int* M){
    assert(sizeA+sizeB <= 1024);
    
    int i = threadIdx.x;
    int Kx, Ky, Px, Py;
    if (i >= sizeA) {
        Kx = i - sizeA;
        Ky = sizeA;
        Px = sizeA;
        Py = i - sizeA;
    } else {
        Kx = 0;
        Ky = i;
        Px = i;
        Py = 0;
    }

    while (1) {
        int offset = abs(Ky - Py) / 2;
        int Qx = Kx + offset;
        int Qy = Ky - offset;

        if (Qy >= 0 && Qx <= sizeB && (Qy == sizeA || Qx == 0 || A[Qy] > B[Qx - 1])) {
            if (Qx == sizeB || Qy == 0 || A[Qy - 1] <= B[Qx]) {
                if (Qy < sizeA && (Qx == sizeB || A[Qy] <= B[Qx])) {
                    M[i] = A[Qy];
                } else {
                    M[i] = B[Qx];
                }
                break;
            } else {
                Kx = Qx + 1;
                Ky = Qy - 1;
            }
        } else {
            Px = Qx - 1;
            Py = Qy + 1;
        }
    }
}

int main(){
    int *a, *b, *m, *aGPU, *bGPU, *mGPU, sizeA, sizeB;
    float TimeVar;
    hipEvent_t start, stop;
    testCUDA(hipEventCreate(&start));
    testCUDA(hipEventCreate(&stop));

    sizeA = sizeB = 120;
    int sizeM = sizeA + sizeB;

    // Initialisez le générateur de nombres aléatoires
    srand(time(NULL));

    printf("Generating A and B :\n");
    a = generateSortedRandomArray(sizeA);
    b = generateSortedRandomArray(sizeB);
    m = (int*)malloc(sizeM * sizeof(int)); // Allocation de mémoire pour m
    if (!m) {
        printf("Erreur d'allocation de mémoire pour m.\n");
        exit(1);
    }
    if (isSorted(a,sizeA)){
        printf("A is sorted of size = %d\n",sizeA);
    }
    if (isSorted(b,sizeB)){
        printf("B is sorted of size = %d\n",sizeB);
    }

	testCUDA(hipMalloc(&aGPU,sizeA*sizeof(int)));
	testCUDA(hipMalloc(&bGPU,sizeB*sizeof(int)));
	testCUDA(hipMalloc(&mGPU,sizeM*sizeof(int)));

	testCUDA(hipEventRecord(start,0));

    testCUDA(hipMemcpy(aGPU, a, sizeA*sizeof(int),	hipMemcpyHostToDevice)); 
    testCUDA(hipMemcpy(bGPU, b, sizeB*sizeof(int),	hipMemcpyHostToDevice));
    printf("Sorting A and B to M ...\n"); 
    mergeSmall_k<<< 1,sizeM >>>(aGPU,sizeA,bGPU,sizeB,mGPU);

    testCUDA(hipMemcpy(m, mGPU, sizeM*sizeof(int),	hipMemcpyDeviceToHost));
    if (isSorted(m,sizeM)){
        printf("M is sorted\n");
    }
	
	testCUDA(hipEventRecord(stop,0));
	testCUDA(hipEventSynchronize(stop));
	testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
	testCUDA(hipEventDestroy(start));
	testCUDA(hipEventDestroy(stop));
	testCUDA(hipFree(aGPU));
	testCUDA(hipFree(bGPU));
	testCUDA(hipFree(mGPU));
	free(a);	
    free(b);	
    free(m); // Libération de la mémoire pour m

    printf("Processing time when using malloc: %f s\n", 0.001f * TimeVar);
    
    return 0;
}